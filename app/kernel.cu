#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <ctime>
#include <cmath>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <cstdlib> 
#include <hiprand.h>
using namespace std;

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <random>

#define Npoints 40000

__global__ void setup_kernel(hiprandState* state, unsigned long seed)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, i, 0, &state[i]);
}

__global__ void generate(hiprandState* globalState, float* devRez)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localState = globalState[i];
    double x = hiprand_normal_double(&localState);
    double y = hiprand_uniform_double(&localState);
    if ((x*x + y*y) <= 1) {
        devRez[i] = 1.0;
    }
    globalState[i] = localState;
    
}

int main(int argc, char** argv)
{
    hiprandState* devStates;
    float* rez = new float[Npoints];
    float* devRez;

    hipSetDevice(0);
    hipMalloc(&devStates, Npoints * sizeof(hiprandState));
    setup_kernel << < Npoints/1024 + 1, 1024 >> > (devStates, time(0));

    hipMalloc(&devRez, Npoints * sizeof(*rez));
    float gpuTime;

    hipEvent_t gpu_start, gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start, 0);
    
    generate << < Npoints / 1024 + 1, 1024 >> > (devStates, devRez);

    hipEventRecord(gpu_stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&gpuTime, gpu_start, gpu_stop);
    hipMemcpy(rez, devRez, Npoints * sizeof(*rez), hipMemcpyDeviceToHost);
    double NincircleGpu = 0;
    cout << NincircleGpu << endl;
    for (int i = 0; i < Npoints; i++)
    {
        if (rez[i] == 1) {
            NincircleGpu++;
        }
    }
    cout << NincircleGpu << endl;
   double piGPU = (double)(NincircleGpu * 4.0 / (double)Npoints);
   printf("PI gpu: %.5f \n", piGPU);
   hipFree(devRez);
   hipFree(devStates);

   double Nincircle = 0;
   double x;
   double y;
   unsigned int cpu_start = clock();
   srand(time(0));
       for (int i = 0; i < Npoints; i++) {
           x = 0.01 * (rand() % 101);
           y = 0.01 * (rand() % 101);

           if (x*x + y*y <= 1) {
               Nincircle++;
           }
       }
   unsigned int cpu_end = clock();
   double pi = (double)(Nincircle * 4.0 / (double)Npoints);
   
   printf("PI cpu %.5f \n", pi);

   printf("gpu time= %.5f seconds\n", gpuTime / CLOCKS_PER_SEC);
   printf("cpu time= %.5f seconds\n", (double)(cpu_end - cpu_start) / CLOCKS_PER_SEC);
   delete rez;
    return 0;
}



